#include "chronoGPU.hpp"
#include "commonCUDA.hpp"
#include <iostream>

ChronoGPU::ChronoGPU()
	: _started(false)
{
	HANDLE_ERROR(hipEventCreate(&_start));
	HANDLE_ERROR(hipEventCreate(&_end));
}

ChronoGPU::~ChronoGPU()
{
	if (_started)
	{
		std::cerr << "ChronoGPU::~ChronoGPU(): chrono wasn't turned off!" << std::endl;
	}
	HANDLE_ERROR(hipEventDestroy(_start));
	HANDLE_ERROR(hipEventDestroy(_end));
}

void ChronoGPU::start()
{
	if (!_started)
	{
		HANDLE_ERROR(hipEventRecord(_start, 0));
		_started = true;
	}
	else
	{
		std::cerr << "ChronoGPU::start(): chrono has already started!" << std::endl;
	}
}

void ChronoGPU::stop()
{
	if (_started)
	{
		HANDLE_ERROR(hipEventRecord(_end, 0));
		HANDLE_ERROR(hipEventSynchronize(_end));
		_started = false;
	}
	else
	{
		std::cerr << "ChronoGPU::stop(): chrono hadn't started!" << std::endl;
	}
}

float ChronoGPU::elapsedTime()
{
	float time = 0.f;
	if (_started)
	{
		std::cerr << "ChronoGPU::elapsedTime(): chrono is still running!" << std::endl;
	}
	else
	{
		HANDLE_ERROR(hipEventElapsedTime(&time, _start, _end));
	}
	return time;
}
